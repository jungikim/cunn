#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "common.h"

template <typename T, typename AccumT>
__global__ void cunn_SVDLinear_updateFullView_kernel(
    const int nthreads,
    long *indices,
    T *z,
    T *B,
    T *h,
    T *bias,
    const int N,
    const int batchSize,
    const int V,
    const int D)
{
  CUDA_KERNEL_LOOP(index, nthreads) {

    const int nIdx = index / batchSize;
    const int bIdx = index % batchSize;

    long vIdx = indices[nIdx * batchSize + bIdx] - 1;
    AccumT dot = AccumT(0);
    for(int dIdx = 0 ; dIdx < D ; dIdx ++)
        dot += THCNumerics<T>::mul(B[vIdx * D + dIdx],
                                   h[dIdx * batchSize + bIdx]);

    if (bias)
      dot += bias[vIdx];

    z[vIdx * batchSize + bIdx] = ScalarConvert<AccumT, T>::to(dot);

    index += blockDim.x;
  }
}

#include "generic/SVDLinear.cu"
#include "THCGenerateHalfType.h"
#include "generic/SVDLinear.cu"
#include "THCGenerateFloatType.h"
#include "generic/SVDLinear.cu"
#include "THCGenerateDoubleType.h"
