
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SVDLinear.cu"
#else

void THNN_(SVDLinear_updateFullView)(
           THCState *state,
           THCIndexTensor *indices,
           THCTensor *z,
           THCTensor *B,
           THCTensor *h,
           THCTensor *bias)
{
  THCUNN_assertSameGPU(state, 5, indices, z, B, h, bias);

  indices = THCIndexTensor_(newContiguous)(state, indices);
  z = THCTensor_(newContiguous)(state, z);
  B = THCTensor_(newContiguous)(state, B);
  h = THCTensor_(newContiguous)(state, h);
  if (bias)
    bias = THCTensor_(newContiguous)(state, bias);

  long N = THCIndexTensor_(size)(state, indices, 0);
  long V = THCTensor_(size)(state, z, 0);
  long D = THCTensor_(size)(state, h, 0);

  int count = THCIndexTensor_(nElement)(state, indices);

  long batchSize = 1;
  if (THCIndexTensor_(nDimension)(state, indices) > 1)
    batchSize = THCIndexTensor_(size)(state, indices, 1);

  cunn_SVDLinear_updateFullView_kernel<real, accreal>
    <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count,
      THCIndexTensor_(data)(state, indices),
      THCTensor_(data)(state, z),
      THCTensor_(data)(state, B),
      THCTensor_(data)(state, h),
      THCTensor_(data)(state, bias),
      N,
      batchSize,
      V,
      D
  );
}

#endif
